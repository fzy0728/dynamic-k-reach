#include "hip/hip_runtime.h"
#include "GPUKReach.h"

using namespace std;

void gpu_bfs(const GPUKReach &kReach, vertex_t s, std::vector<distance_t> &dist) {
    distance_t* distance_data = kReach.d_distance_.data();
    int* visited_bits_data = kReach.d_visited_bits_.data();
    const degree_t* vertices = kReach.graph_.gpu_vertices();
    const vertex_t* edges = kReach.graph_.gpu_edges();

    hipMemset(distance_data, 0x3f, kReach.d_distance_.size() * sizeof(distance_t));
    hipMemset(visited_bits_data, 0, kReach.d_visited_bits_.size() * sizeof(int));

    mgpu::transform([=]MGPU_DEVICE(int index) {
    visited_bits_data[s / 32] = 1 << (31 & s);
    distance_data[s] = 0;
    }, 1, mgpu_context);

  workload_t wl;
  wl.count = kReach.graph_.successors(s).size();
  wl.num_segments = 1;
  vector<int> edge_indices_host = { (int) kReach.graph_.vertices().at(s) };
  wl.segments = mgpu::fill<int>(0, 1, mgpu_context);
  wl.edge_indices = mgpu::to_mem(edge_indices_host, mgpu_context);
  for (int cur_level = 0; cur_level < kReach.k_ && wl.num_segments; ++cur_level) {
      // Create a dynamic work-creation engine.
    auto engine = mgpu::expt::lbs_workcreate(wl.count, wl.segments.data(), 
        wl.num_segments, mgpu_context);

    // The upsweep attempts atomicOr. If it succeeds, return the number of 
    // edges for that vertex.
    auto wl2_count = engine.upsweep(
        [=]MGPU_DEVICE(int index, int seg, int rank, mgpu::tuple<int> desc) {
        int count = 0;
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int mask = 1<< (31 & neighbor);
        if (0 == (mask & atomicOr(visited_bits_data + neighbor / 32, mask))) {
            count = vertices[neighbor + 1] - vertices[neighbor];
            distance_data[neighbor] = cur_level + 1;
        }
        return count;
        }, mgpu::make_tuple(wl.edge_indices.data())
    );

    // The downsweep streams out the new edge pointers.
    mgpu::mem_t<int> edge_indices(wl2_count.num_segments, mgpu_context);
    mgpu::mem_t<int> segments = engine.downsweep(
        [=]MGPU_DEVICE(int dest_seg, int index, int seg, int rank, 
        mgpu::tuple<int> desc, int* out_edge_indices) {
        // Return the same count as before and store output segment-specific
        // data using dest_seg.
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int begin = vertices[neighbor];
        int end = vertices[neighbor + 1];

        // Store the pointer into the edges array for the new work segment.
        out_edge_indices[dest_seg] = begin;

        return end - begin;
        }, mgpu::make_tuple(wl.edge_indices.data()), edge_indices.data()
    );

    // Update the workload.
    wl.count = wl2_count.count;
    wl.num_segments = wl2_count.num_segments;
    wl.segments = std::move(segments);
    wl.edge_indices = std::move(edge_indices);
  }
  mgpu::dtoh(dist, distance_data, dist.size());
}

GPUKReach::GPUKReach(const GPUGraph &graph, distance_t k)
        : graph_(graph), k_(k), degree_(graph.num_vertices()), queue_(graph.num_vertices()), 
          d_distance_(graph.num_vertices(), mgpu_context), d_visited_bits_(mgpu::div_up(graph.num_vertices(), 32), mgpu_context){
    vector<pair<degree_t, vertex_t>> quedeg_temp_;
    quedeg_temp_.reserve(graph.num_vertices());
    DegreeQueue(less<pair<degree_t, vertex_t>>(), move(quedeg_temp_)).swap(quedeg_);
}

void GPUKReach::construct() {
    index_.clear();
    set_degree();
    for (vertex_t i = 0; i < graph_.num_vertices(); ++i){
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty()){
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        index_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
        construct_bfs(cur, index_.at(cur));
        cover(cur);
    }
}

void GPUKReach::cover(vertex_t v) {
    degree_.at(v) = 0;
    for (const auto &i : graph_.successors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
    for (const auto &i : graph_.predecessors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
}

void GPUKReach::set_degree() {
    copy(graph_.degree().begin(), graph_.degree().end(), degree_.begin());
}

void GPUKReach::construct_bfs(vertex_t s, std::vector<distance_t> &dist) {
    gpu_bfs(*this, s, dist);
}

distance_t GPUKReach::distance(vertex_t s, vertex_t t) const {
    return index_.at(s).at(t);
}

bool GPUKReach::indexed(vertex_t v) const {
    return index_.find(v) != index_.end();
}

bool GPUKReach::query(vertex_t s, vertex_t t) const {
    if (s == t && s < graph_.num_vertices()){
        return true;
    }
    if (indexed(s) && indexed(t)){
        return distance(s, t) <= k_;
    }
    else if (indexed(s) && !indexed(t)){
        for (const auto &v : graph_.predecessors(t)){
            if (distance(s, v) + 1 <= k_){
                return true;
            }
        }
    }
    else if (!indexed(s) && indexed(t)){
        for (const auto &v : graph_.successors(s)){
            if (distance(v, t) + 1 <= k_){
                return true;
            }
        }
    }
    else {
        for (const auto &u : graph_.successors(s)){
            for (const auto &v : graph_.predecessors(t)){
                if (distance(u, v) + 2 <= k_){
                    return true;
                }
            }
        }
    }
    return false;
}

void GPUKReach::insert_edge(vertex_t s, vertex_t t) {
    (void) s;
    (void) t;
    construct();
}

void GPUKReach::remove_edge(vertex_t s, vertex_t t) {
    (void) s;
    (void) t;
    construct();
}

void GPUKReach::remove_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    (void) v;
    (void) out;
    (void) in;
    construct();
}

void GPUKReach::insert_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    (void) v;
    (void) out;
    (void) in;
    construct();
}


