#include "hip/hip_runtime.h"
#include "DynamicGPUKReach.h"

__device__ char atomicMinChar(char* address, char val) {
    unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
    unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
    unsigned int sel = selectors[(size_t)address & 3];
    unsigned int old, assumed, min_, new_;

    old = *base_address;
    do {
        assumed = old;
        min_ = min(val, (char)__byte_perm(old, 0, ((size_t)address & 3) | 0x4440));
        new_ = __byte_perm(old, min_, sel);
        if (new_ == old){
            break;
        }
        old = atomicCAS(base_address, assumed, new_);
    } while (assumed != old);

    return old;
}

using namespace std;

void gpu_bfs(const DynamicGPUKReach &kReach, vertex_t s, std::vector<distance_t> &dist) {
    distance_t* distance_data = kReach.d_distance_.data();
    int* visited_bits_data = kReach.d_visited_bits_.data();
    const degree_t* vertices = kReach.graph_.gpu_vertices();
    const vertex_t* edges = kReach.graph_.gpu_edges();

    hipMemset(distance_data, 0x3f, kReach.d_distance_.size() * sizeof(distance_t));
    hipMemset(visited_bits_data, 0, kReach.d_visited_bits_.size() * sizeof(int));

    mgpu::transform([=]MGPU_DEVICE(int index) {
    visited_bits_data[s / 32] = 1 << (31 & s);
    distance_data[s] = 0;
    }, 1, mgpu_context);

  workload_t wl;
  wl.count = kReach.graph_.successors(s).size();
  wl.num_segments = 1;
  vector<int> edge_indices_host = { (int) kReach.graph_.vertices().at(s) };
  wl.segments = mgpu::fill<int>(0, 1, mgpu_context);
  wl.edge_indices = mgpu::to_mem(edge_indices_host, mgpu_context);
  for (int cur_level = 0; cur_level < kReach.k_ && wl.num_segments; ++cur_level) {
      // Create a dynamic work-creation engine.
    auto engine = mgpu::expt::lbs_workcreate(wl.count, wl.segments.data(), 
        wl.num_segments, mgpu_context);

    // The upsweep attempts atomicOr. If it succeeds, return the number of 
    // edges for that vertex.
    auto wl2_count = engine.upsweep(
        [=]MGPU_DEVICE(int index, int seg, int rank, mgpu::tuple<int> desc) {
        int count = 0;
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int mask = 1<< (31 & neighbor);
        if (0 == (mask & atomicOr(visited_bits_data + neighbor / 32, mask))) {
            count = vertices[neighbor + 1] - vertices[neighbor];
            distance_data[neighbor] = cur_level + 1;
        }
        return count;
        }, mgpu::make_tuple(wl.edge_indices.data())
    );

    // The downsweep streams out the new edge pointers.
    mgpu::mem_t<int> edge_indices(wl2_count.num_segments, mgpu_context);
    mgpu::mem_t<int> segments = engine.downsweep(
        [=]MGPU_DEVICE(int dest_seg, int index, int seg, int rank, 
        mgpu::tuple<int> desc, int* out_edge_indices) {
        // Return the same count as before and store output segment-specific
        // data using dest_seg.
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int begin = vertices[neighbor];
        int end = vertices[neighbor + 1];

        // Store the pointer into the edges array for the new work segment.
        out_edge_indices[dest_seg] = begin;

        return end - begin;
        }, mgpu::make_tuple(wl.edge_indices.data()), edge_indices.data()
    );

    // Update the workload.
    wl.count = wl2_count.count;
    wl.num_segments = wl2_count.num_segments;
    wl.segments = std::move(segments);
    wl.edge_indices = std::move(edge_indices);
  }
  mgpu::dtoh(dist, distance_data, dist.size());
}

void gpu_resume_bfs(const DynamicGPUKReach &kReach, vertex_t s, std::vector<distance_t> &dist) {
    distance_t* distance_data = kReach.d_distance_.data();
    int* visited_bits_data = kReach.d_visited_bits_.data();
    const degree_t* vertices = kReach.graph_.gpu_vertices();
    const vertex_t* edges = kReach.graph_.gpu_edges();

    mgpu::htod(distance_data, dist.data(), dist.size());
    hipMemset(visited_bits_data, 0, kReach.d_visited_bits_.size() * sizeof(int));

    mgpu::transform([=]MGPU_DEVICE(int index) {
        if (distance_data[index] < 0x3f) {
            atomicOr(visited_bits_data + index / 32, 1 << (31 & index));
        }
    }, dist.size(), mgpu_context);

  workload_t wl;
  wl.count = kReach.graph_.successors(s).size();
  wl.num_segments = 1;
  vector<int> edge_indices_host = { (int) kReach.graph_.vertices().at(s) };
  wl.segments = mgpu::fill<int>(0, 1, mgpu_context);
  wl.edge_indices = mgpu::to_mem(edge_indices_host, mgpu_context);
  for (int cur_level = dist.at(s); cur_level < kReach.k_ && wl.num_segments; ++cur_level) {
      // Create a dynamic work-creation engine.
    auto engine = mgpu::expt::lbs_workcreate(wl.count, wl.segments.data(), 
        wl.num_segments, mgpu_context);

    // The upsweep attempts atomicOr. If it succeeds, return the number of 
    // edges for that vertex.
    auto wl2_count = engine.upsweep(
        [=]MGPU_DEVICE(int index, int seg, int rank, mgpu::tuple<int> desc) {
        int count = 0;
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int mask = 1<< (31 & neighbor);
        if (0 == (mask & atomicOr(visited_bits_data + neighbor / 32, mask))) {
            count = vertices[neighbor + 1] - vertices[neighbor];
            distance_data[neighbor] = cur_level + 1;
        }
        else if (cur_level + 1 < atomicMinChar((char*)distance_data + neighbor, cur_level + 1)){
            count = vertices[neighbor + 1] - vertices[neighbor];
        }
        return count;
        }, mgpu::make_tuple(wl.edge_indices.data())
    );

    // The downsweep streams out the new edge pointers.
    mgpu::mem_t<int> edge_indices(wl2_count.num_segments, mgpu_context);
    mgpu::mem_t<int> segments = engine.downsweep(
        [=]MGPU_DEVICE(int dest_seg, int index, int seg, int rank, 
        mgpu::tuple<int> desc, int* out_edge_indices) {
        // Return the same count as before and store output segment-specific
        // data using dest_seg.
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int begin = vertices[neighbor];
        int end = vertices[neighbor + 1];

        // Store the pointer into the edges array for the new work segment.
        out_edge_indices[dest_seg] = begin;

        return end - begin;
        }, mgpu::make_tuple(wl.edge_indices.data()), edge_indices.data()
    );

    // Update the workload.
    wl.count = wl2_count.count;
    wl.num_segments = wl2_count.num_segments;
    wl.segments = std::move(segments);
    wl.edge_indices = std::move(edge_indices);
  }
  mgpu::dtoh(dist, distance_data, dist.size());
}

DynamicGPUKReach::DynamicGPUKReach(const GPUGraph &graph, distance_t k)
        : graph_(graph), k_(k), degree_(graph.num_vertices()), queue_(graph.num_vertices()), 
          d_distance_(graph.num_vertices(), mgpu_context), d_visited_bits_(mgpu::div_up(graph.num_vertices(), 32), mgpu_context){
    vector<pair<degree_t, vertex_t>> quedeg_temp_;
    quedeg_temp_.reserve(graph.num_vertices());
    DegreeQueue(less<pair<degree_t, vertex_t>>(), move(quedeg_temp_)).swap(quedeg_);
    vector<pair<distance_t, vertex_t>> quedist_temp_;
    quedist_temp_.reserve(graph.num_vertices());
    DistanceQueue(greater<pair<distance_t, vertex_t>>(), move(quedist_temp_)).swap(quedist_);
}

void DynamicGPUKReach::construct() {
    index_.clear();
    set_degree();
    for (vertex_t i = 0; i < graph_.num_vertices(); ++i){
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty()){
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        index_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
        construct_bfs(cur, index_.at(cur));
        cover(cur);
    }
}

bool DynamicGPUKReach::query(vertex_t s, vertex_t t) const {
    if (s == t && s < graph_.num_vertices()){
        return true;
    }
    if (indexed(s) && indexed(t)){
        return distance(s, t) <= k_;
    }
    else if (indexed(s) && !indexed(t)){
        for (const auto &v : graph_.predecessors(t)){
            if (distance(s, v) + 1 <= k_){
                return true;
            }
        }
    }
    else if (!indexed(s) && indexed(t)){
        for (const auto &v : graph_.successors(s)){
            if (distance(v, t) + 1 <= k_){
                return true;
            }
        }
    }
    else {
        for (const auto &u : graph_.successors(s)){
            for (const auto &v : graph_.predecessors(t)){
                if (distance(u, v) + 2 <= k_){
                    return true;
                }
            }
        }
    }
    return false;
}

bool DynamicGPUKReach::indexed(vertex_t v) const {
    return index_.find(v) != index_.end();
}

distance_t DynamicGPUKReach::distance(vertex_t s, vertex_t t) const {
    return index_.at(s).at(t);
}

void DynamicGPUKReach::set_degree() {
    copy(graph_.degree().begin(), graph_.degree().end(), degree_.begin());
}

void DynamicGPUKReach::cover(vertex_t v) {
    degree_.at(v) = 0;
    for (const auto &i : graph_.successors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
    for (const auto &i : graph_.predecessors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
}

void DynamicGPUKReach::construct_bfs(vertex_t s, std::vector<distance_t> &dist) {
    gpu_bfs(*this, s, dist);
}

void DynamicGPUKReach::insert_edge(vertex_t s, vertex_t t) {
    index_temp_.clear();
    swap(index_, index_temp_);
    set_degree();
    for (vertex_t i = 0; i < graph_.num_vertices(); ++i){
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty()){
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (index_temp_.find(cur) != index_temp_.end()){
            index_[cur] = move(index_temp_.at(cur));
            update_insert(s, t, index_.at(cur));
        }
        else {
            index_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
            construct_bfs(cur, index_.at(cur));
        }
        index_temp_.erase(cur);
        cover(cur);
    }
}

void DynamicGPUKReach::update_insert(vertex_t s, vertex_t t, std::vector<distance_t> &dist) {
    if (dist.at(s) >= k_ || dist.at(t) <= dist.at(s) + 1) {
        return;
    }
    dist.at(t) = dist.at(s) + 1;
    resume_bfs(t, dist);
}

void DynamicGPUKReach::resume_bfs(vertex_t s, std::vector<distance_t> &dist) {  
    gpu_resume_bfs(*this, s, dist);
}

void DynamicGPUKReach::insert_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    index_temp_.clear();
    swap(index_, index_temp_);
    set_degree();
    for (vertex_t i = 0; i < graph_.num_vertices(); ++i){
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty()){
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (index_temp_.find(cur) != index_temp_.end()){
            index_[cur] = move(index_temp_.at(cur));
            update_insert(v, out, in, index_.at(cur));
        }
        else {
            index_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
            construct_bfs(cur, index_.at(cur));
        }
        index_temp_.erase(cur);
        cover(cur);
    }
}

void DynamicGPUKReach::update_insert(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in,
                                  std::vector<distance_t> &dist) {
    auto d = dist.at(v);
    for (const auto &s : in){
        if (dist.at(s) < k_ && dist.at(s) + 1 < dist.at(v)){
            dist.at(v) = dist.at(s) + 1;
        }
    }

    if (dist.at(v) >= k_){
        return;
    }

    if (dist.at(v) < d){
        resume_bfs(v, dist);
    }
    else {
        for (const auto &t : out) {
            if (dist.at(v) + 1 < dist.at(t)){
                dist.at(t) = dist.at(v) + 1;
                resume_bfs(t, dist);
            }
        }
    }
}

void DynamicGPUKReach::remove_edge(vertex_t s, vertex_t t) {
    index_temp_.clear();
    swap(index_, index_temp_);
    set_degree();
    for (vertex_t i = 0; i < graph_.num_vertices(); ++i){
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty()){
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (index_temp_.find(cur) != index_temp_.end()){
            index_[cur] = move(index_temp_.at(cur));
            update_remove(cur, s, t, index_.at(cur));
        }
        else {
            index_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
            construct_bfs(cur, index_.at(cur));
        }
        index_temp_.erase(cur);
        cover(cur);
    }
}

void DynamicGPUKReach::update_remove(vertex_t r, vertex_t s, vertex_t t, std::vector<distance_t> &dist) {
    if (dist.at(s) >= k_ || dist.at(t) > k_ || dist.at(s) + 1 != dist.at(t)) {
        return;
    }
    gpu_bfs(*this, r, dist);
}

void DynamicGPUKReach::remove_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    (void) in;
    index_temp_.clear();
    swap(index_, index_temp_);
    set_degree();
    for (vertex_t i = 0; i < graph_.num_vertices(); ++i){
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty()){
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (index_temp_.find(cur) != index_temp_.end()){
            index_[cur] = move(index_temp_.at(cur));
            update_remove(cur, v, out, index_.at(cur));
        }
        else {
            index_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
            construct_bfs(cur, index_.at(cur));
        }
        index_temp_.erase(cur);
        cover(cur);
    }
}

void DynamicGPUKReach::update_remove(vertex_t r, vertex_t v, const std::vector<vertex_t> &out, std::vector<distance_t> &dist) {
    if (dist.at(v) > k_) {
        return;
    }    
    gpu_bfs(*this, r, dist);
}