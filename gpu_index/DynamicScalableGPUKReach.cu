#include "hip/hip_runtime.h"
#include "DynamicScalableGPUKReach.h"

__device__ char atomicMinCharS(char* address, char val) {
    unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
    unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
    unsigned int sel = selectors[(size_t)address & 3];
    unsigned int old, assumed, min_, new_;

    old = *base_address;
    do {
        assumed = old;
        min_ = min(val, (char)__byte_perm(old, 0, ((size_t)address & 3) | 0x4440));
        new_ = __byte_perm(old, min_, sel);
        if (new_ == old){
            break;
        }
        old = atomicCAS(base_address, assumed, new_);
    } while (assumed != old);

    return old;
}

using namespace std;

void gpu_bfs(const DynamicScalableGPUKReach::DynamicPartialIndex &kReach, vertex_t s, const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
    const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    distance_t* distance_data = kReach.d_distance_.data();
    int* visited_bits_data = kReach.d_visited_bits_.data();
    auto d1_bits_data = kReach.d_d1_bits_.data();
    auto isD2 = kReach.isD2_;

    hipMemset(distance_data, 0x3f, kReach.d_distance_.size() * sizeof(distance_t));
    hipMemset(visited_bits_data, 0, kReach.d_visited_bits_.size() * sizeof(int));

    mgpu::transform([=]MGPU_DEVICE(int index) {
    visited_bits_data[s / 32] = 1 << (31 & s);
    distance_data[s] = 0;
    }, 1, mgpu_context);

  workload_t wl;
  wl.count = h_vertices.at(s + 1) - h_vertices.at(s);
  wl.num_segments = 1;
  vector<int> edge_indices_host = { (int) h_vertices.at(s) };
  wl.segments = mgpu::fill<int>(0, 1, mgpu_context);
  wl.edge_indices = mgpu::to_mem(edge_indices_host, mgpu_context);
  for (int cur_level = 0; cur_level < kReach.k_ && wl.num_segments; ++cur_level) {
      // Create a dynamic work-creation engine.
    auto engine = mgpu::expt::lbs_workcreate(wl.count, wl.segments.data(), 
        wl.num_segments, mgpu_context);

    // The upsweep attempts atomicOr. If it succeeds, return the number of 
    // edges for that vertex.
    auto wl2_count = engine.upsweep(
        [=]MGPU_DEVICE(int index, int seg, int rank, mgpu::tuple<int> desc) {
        int count = 0;
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int mask = 1<< (31 & neighbor);
        if (0 == (mask & atomicOr(visited_bits_data + neighbor / 32, mask))) {
            if (!isD2 || !(mask & d1_bits_data[neighbor / 32])){
                count = vertices[neighbor + 1] - vertices[neighbor];
            }
            distance_data[neighbor] = cur_level + 1;
        }
        return count;
        }, mgpu::make_tuple(wl.edge_indices.data())
    );

    // The downsweep streams out the new edge pointers.
    mgpu::mem_t<int> edge_indices(wl2_count.num_segments, mgpu_context);
    mgpu::mem_t<int> segments = engine.downsweep(
        [=]MGPU_DEVICE(int dest_seg, int index, int seg, int rank, 
        mgpu::tuple<int> desc, int* out_edge_indices) {
        // Return the same count as before and store output segment-specific
        // data using dest_seg.
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int begin = vertices[neighbor];
        int end = vertices[neighbor + 1];

        // Store the pointer into the edges array for the new work segment.
        out_edge_indices[dest_seg] = begin;

        return end - begin;
        }, mgpu::make_tuple(wl.edge_indices.data()), edge_indices.data()
    );

    // Update the workload.
    wl.count = wl2_count.count;
    wl.num_segments = wl2_count.num_segments;
    wl.segments = std::move(segments);
    wl.edge_indices = std::move(edge_indices);
  }
  mgpu::dtoh(dist, distance_data, dist.size());
}

void gpu_set_d1(const DynamicScalableGPUKReach::DynamicPartialIndex &kReach) {
    auto d1_bits_data = kReach.d_d1_bits_.data();
    auto d1_data = kReach.d_d1_.data();
    mgpu::htod(d1_data, kReach.queue_.data(), kReach.parent_.D1_.succ_.size());
    hipMemset(d1_bits_data, 0, kReach.d_d1_bits_.size() * sizeof(int));
    mgpu::transform([=]MGPU_DEVICE(int index) {
        vertex_t vertex = d1_data[index];
        atomicOr(d1_bits_data + vertex / 32, 1 << (31 & vertex));
    }, kReach.parent_.D1_.succ_.size(), mgpu_context);
}

void gpu_resume_bfs(const DynamicScalableGPUKReach::DynamicPartialIndex &kReach, vertex_t s, const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
    const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    distance_t* distance_data = kReach.d_distance_.data();
    int* visited_bits_data = kReach.d_visited_bits_.data();
    auto d1_bits_data = kReach.d_d1_bits_.data();
    auto isD2 = kReach.isD2_;

    mgpu::htod(distance_data, dist.data(), dist.size());
    hipMemset(visited_bits_data, 0, kReach.d_visited_bits_.size() * sizeof(int));

    mgpu::transform([=]MGPU_DEVICE(int index) {
        if (distance_data[index] < 0x3f) {
            atomicOr(visited_bits_data + index / 32, 1 << (31 & index));
        }
    }, dist.size(), mgpu_context);

  workload_t wl;
  wl.count = h_vertices.at(s + 1) - h_vertices.at(s);
  wl.num_segments = 1;
  vector<int> edge_indices_host = { (int) h_vertices.at(s) };
  wl.segments = mgpu::fill<int>(0, 1, mgpu_context);
  wl.edge_indices = mgpu::to_mem(edge_indices_host, mgpu_context);
  for (int cur_level = dist.at(s); cur_level < kReach.k_ && wl.num_segments; ++cur_level) {
      // Create a dynamic work-creation engine.
    auto engine = mgpu::expt::lbs_workcreate(wl.count, wl.segments.data(), 
        wl.num_segments, mgpu_context);

    // The upsweep attempts atomicOr. If it succeeds, return the number of 
    // edges for that vertex.
    auto wl2_count = engine.upsweep(
        [=]MGPU_DEVICE(int index, int seg, int rank, mgpu::tuple<int> desc) {
        int count = 0;
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int mask = 1<< (31 & neighbor);
        if (0 == (mask & atomicOr(visited_bits_data + neighbor / 32, mask))) {
            if (!isD2 || !(mask & d1_bits_data[neighbor / 32])){
                count = vertices[neighbor + 1] - vertices[neighbor];
            }
            distance_data[neighbor] = cur_level + 1;
        }
        else if (cur_level + 1 < atomicMinCharS((char*)distance_data + neighbor, cur_level + 1)){
            if (!isD2 || !(mask & d1_bits_data[neighbor / 32])){
                count = vertices[neighbor + 1] - vertices[neighbor];
            }
        }
        return count;
        }, mgpu::make_tuple(wl.edge_indices.data())
    );

    // The downsweep streams out the new edge pointers.
    mgpu::mem_t<int> edge_indices(wl2_count.num_segments, mgpu_context);
    mgpu::mem_t<int> segments = engine.downsweep(
        [=]MGPU_DEVICE(int dest_seg, int index, int seg, int rank, 
        mgpu::tuple<int> desc, int* out_edge_indices) {
        // Return the same count as before and store output segment-specific
        // data using dest_seg.
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int begin = vertices[neighbor];
        int end = vertices[neighbor + 1];

        // Store the pointer into the edges array for the new work segment.
        out_edge_indices[dest_seg] = begin;

        return end - begin;
        }, mgpu::make_tuple(wl.edge_indices.data()), edge_indices.data()
    );

    // Update the workload.
    wl.count = wl2_count.count;
    wl.num_segments = wl2_count.num_segments;
    wl.segments = std::move(segments);
    wl.edge_indices = std::move(edge_indices);
  }
  mgpu::dtoh(dist, distance_data, dist.size());
}


void DynamicScalableGPUKReach::DynamicPartialIndex::resume_bfs(vertex_t s, const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
        const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    gpu_resume_bfs(*this, s, h_vertices, h_edges, vertices, edges, dist);
}

void DynamicScalableGPUKReach::DynamicPartialIndex::update_insert(vertex_t s, vertex_t t,
        const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
        const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    if (dist.at(s) >= k_ || dist.at(t) <= dist.at(s) + 1) {
        return;
    }
    dist.at(t) = dist.at(s) + 1;
    resume_bfs(t, h_vertices, h_edges, vertices, edges, dist);
}

void DynamicScalableGPUKReach::DynamicPartialIndex::insert_edge(vertex_t s, vertex_t t) {
    succ_temp_.clear();
    swap(succ_, succ_temp_);
    pred_temp_.clear();
    swap(pred_, pred_temp_);
    set_degree();
    for (degree_t i = 0; i < graph_.num_vertices(); ++i) {
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty() && succ_.size() != budget_) {
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (!isD2_) { // is D1
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                update_insert(s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                update_insert(t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D2_.indexed(cur)) { // indexed in D2
                succ_[cur] = move(parent_.D2_.succ_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_insert(s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D2_.pred_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_insert(t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        else { // is D2
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_insert(s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_insert(t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D1_.succ_temp_.find(cur) != parent_.D1_.succ_temp_.end()) { // was indexed in D1
                succ_[cur] = move(parent_.D1_.succ_temp_.at(cur));
                update_insert(s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D1_.pred_temp_.at(cur));
                update_insert(t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        succ_temp_.erase(cur);
        pred_temp_.erase(cur);
        update_cover(cur);
    }
    while (!quedeg_.empty()) {
        quedeg_.pop();
    }
}

void DynamicScalableGPUKReach::DynamicPartialIndex::remove_edge(vertex_t s, vertex_t t) {
    succ_temp_.clear();
    swap(succ_, succ_temp_);
    pred_temp_.clear();
    swap(pred_, pred_temp_);
    set_degree();
    for (degree_t i = 0; i < graph_.num_vertices(); ++i) {
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty() && succ_.size() != budget_) {
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (!isD2_) { // is D1
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                update_remove(cur, s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                update_remove(cur, t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D2_.indexed(cur)) { // indexed in D2
                succ_[cur] = move(parent_.D2_.succ_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_remove(cur, s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D2_.pred_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_remove(cur, t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        else { // is D2
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_remove(cur, s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_remove(cur, t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D1_.succ_temp_.find(cur) != parent_.D1_.succ_temp_.end()) { // was indexed in D1
                succ_[cur] = move(parent_.D1_.succ_temp_.at(cur));
                update_remove(cur, s, t, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D1_.pred_temp_.at(cur));
                update_remove(cur, t, s, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        succ_temp_.erase(cur);
        pred_temp_.erase(cur);
        update_cover(cur);
    }
    while (!quedeg_.empty()) {
        quedeg_.pop();
    }
}

void DynamicScalableGPUKReach::DynamicPartialIndex::remove_vertex(vertex_t v, const std::vector<vertex_t> &out,
                                                               const std::vector<vertex_t> &in) {
    succ_temp_.clear();
    swap(succ_, succ_temp_);
    pred_temp_.clear();
    swap(pred_, pred_temp_);
    set_degree();
    for (degree_t i = 0; i < graph_.num_vertices(); ++i) {
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty() && succ_.size() != budget_) {
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (!isD2_) { // is D1
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                update_remove(cur, v, out, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                update_remove(cur, v, in, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D2_.indexed(cur)) { // indexed in D2
                succ_[cur] = move(parent_.D2_.succ_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_remove(cur, v, out, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D2_.pred_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_remove(cur, v, in, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        else { // is D2
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_remove(cur, v, out, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_remove(cur, v, in, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D1_.succ_temp_.find(cur) != parent_.D1_.succ_temp_.end()) { // was indexed in D1
                succ_[cur] = move(parent_.D1_.succ_temp_.at(cur));
                update_remove(cur, v, out, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D1_.pred_temp_.at(cur));
                update_remove(cur, v, in, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        succ_temp_.erase(cur);
        pred_temp_.erase(cur);
        update_cover(cur);
    }
    while (!quedeg_.empty()) {
        quedeg_.pop();
    }
}

DynamicScalableGPUKReach::DynamicPartialIndex::DynamicPartialIndex(DynamicScalableGPUKReach &parent, const GPUGraph &graph,
                                                                distance_t k, uint32_t budget, bool isD2)
        : parent_(parent), graph_(graph), k_(k), budget_(budget), isD2_(isD2),
          quedeg_(parent.quedeg_), degree_(parent.degree_),
          queue_(parent.queue_), quedist_(parent.quedist_), updated_(parent.updated_), 
          d_distance_(parent.d_distance_), d_visited_bits_(parent.d_visited_bits_), d_d1_bits_(parent.d_d1_bits_), d_d1_(parent.d_d1_) {}


void DynamicScalableGPUKReach::DynamicPartialIndex::update_remove(vertex_t r, vertex_t s, vertex_t t,
            const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
            const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    if (dist.at(s) >= k_ || dist.at(t) > k_ || dist.at(s) + 1 != dist.at(t)) {
        return;
    }
    gpu_bfs(*this, r, h_vertices, h_edges, vertices, edges, dist);
}

void DynamicScalableGPUKReach::DynamicPartialIndex::update_remove(vertex_t r, vertex_t v, const std::vector<vertex_t> &out,
            const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
            const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    if (dist.at(v) > k_) {
        return;
    }
    gpu_bfs(*this, r, h_vertices, h_edges, vertices, edges, dist);
}

void DynamicScalableGPUKReach::DynamicPartialIndex::construct() {
    succ_.clear();
    pred_.clear();
    set_degree();
    for (degree_t i = 0; i < graph_.num_vertices(); ++i) {
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty() && succ_.size() != budget_) {
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
        construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
        pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
        construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
        update_cover(cur);
    }
    while (!quedeg_.empty()) {
        quedeg_.pop();
    }
}

bool DynamicScalableGPUKReach::DynamicPartialIndex::indexed(vertex_t v) const {
    return succ_.find(v) != succ_.end();
}

distance_t DynamicScalableGPUKReach::DynamicPartialIndex::distance(vertex_t s, vertex_t t) const {
    return indexed(s) ? succ_.at(s).at(t) : pred_.at(t).at(s);
}

bool DynamicScalableGPUKReach::DynamicPartialIndex::single_intermediate(vertex_t s, vertex_t t) const {
    for (const auto &i : succ_) {
        if (pred_.at(i.first).at(s) + succ_.at(i.first).at(t) <= k_) {
            return true;
        }
    }
    return false;
}

bool DynamicScalableGPUKReach::DynamicPartialIndex::double_intermediate(vertex_t s, vertex_t t) const {
    for (const auto &i : succ_) {
        for (const auto &j : succ_) {
            if (pred_.at(i.first).at(s) + succ_.at(i.first).at(j.first) + succ_.at(j.first).at(t) <= k_) {
                return true;
            }
        }
    }
    return false;
}

void DynamicScalableGPUKReach::DynamicPartialIndex::set_degree() {
    copy(graph_.degree().begin(), graph_.degree().end(), degree_.begin());
    if (isD2_) {
        size_t j = 0;
        for (const auto &i : parent_.D1_.succ_) {
            cover(i.first);
            queue_.at(j++) = i.first;
        }
        gpu_set_d1(*this);
    }
}

void DynamicScalableGPUKReach::DynamicPartialIndex::cover(vertex_t v) {
    degree_.at(v) = 0;
    for (const auto &i : graph_.successors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
    for (const auto &i : graph_.predecessors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
}

void DynamicScalableGPUKReach::DynamicPartialIndex::update_cover(vertex_t v) {
    for (vertex_t j = 0; j < graph_.num_vertices(); ++j) {
        if ((succ_.at(v).at(j) != INF8 || pred_.at(v).at(j) != INF8) &&
            degree_.at(j) > 0) {
            cover(j);
        }
    }
}

void
DynamicScalableGPUKReach::DynamicPartialIndex::construct_bfs(vertex_t s, const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
        const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    gpu_bfs(*this, s, h_vertices, h_edges, vertices, edges, dist);
}

void DynamicScalableGPUKReach::DynamicPartialIndex::insert_vertex(vertex_t v, const std::vector<vertex_t> &out,
                                                               const std::vector<vertex_t> &in) {
    succ_temp_.clear();
    swap(succ_, succ_temp_);
    pred_temp_.clear();
    swap(pred_, pred_temp_);
    set_degree();
    for (degree_t i = 0; i < graph_.num_vertices(); ++i) {
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty() && succ_.size() != budget_) {
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        if (!isD2_) { // is D1
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                update_insert(v, out, in, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                update_insert(v, in, out, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D2_.indexed(cur)) { // indexed in D2
                succ_[cur] = move(parent_.D2_.succ_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_insert(v, out, in, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D2_.pred_.at(cur));
                for (const auto &i : succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_insert(v, in, out, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        else { // is D2
            if (succ_temp_.find(cur) != succ_temp_.end()) { // was indexed
                succ_[cur] = move(succ_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (succ_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                    }
                }
                update_insert(v, out, in, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(pred_temp_.at(cur));
                for (const auto &i : parent_.D1_.succ_temp_) {
                    if (pred_.at(cur).at(i.first) != INF8) {
                        resume_bfs(i.first, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
                    }
                }
                update_insert(v, in, out, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else if (parent_.D1_.succ_temp_.find(cur) != parent_.D1_.succ_temp_.end()) { // was indexed in D1
                succ_[cur] = move(parent_.D1_.succ_temp_.at(cur));
                update_insert(v, out, in, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = move(parent_.D1_.pred_temp_.at(cur));
                update_insert(v, in, out, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
            else { // not indexed before
                succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
                pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
                construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
            }
        }
        succ_temp_.erase(cur);
        pred_temp_.erase(cur);
        update_cover(cur);
    }
    while (!quedeg_.empty()) {
        quedeg_.pop();
    }
}

void DynamicScalableGPUKReach::DynamicPartialIndex::update_insert(vertex_t v, const std::vector<vertex_t> &out,
            const std::vector<vertex_t> &in,
            const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
            const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    auto d = dist.at(v);
    for (const auto &s : in){
        if (dist.at(s) < k_ && dist.at(s) + 1 < dist.at(v)){
            dist.at(v) = dist.at(s) + 1;
        }
    }

    if (dist.at(v) >= k_){
        return;
    }

    if (dist.at(v) < d){
        resume_bfs(v, h_vertices, h_edges, vertices, edges, dist);
    }
    else {
        for (const auto &t : out) {
            if (dist.at(v) + 1 < dist.at(t)){
                dist.at(t) = dist.at(v) + 1;
                resume_bfs(t, h_vertices, h_edges, vertices, edges, dist);
            }
        }
    }
}


void DynamicScalableGPUKReach::insert_edge(vertex_t s, vertex_t t) {
    D1_.insert_edge(s, t);
    D2_.insert_edge(s, t);
}

void DynamicScalableGPUKReach::remove_edge(vertex_t s, vertex_t t) {
    D1_.remove_edge(s, t);
    D2_.remove_edge(s, t);
}

void
DynamicScalableGPUKReach::remove_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    D1_.remove_vertex(v, out, in);
    D2_.remove_vertex(v, out, in);
}

DynamicScalableGPUKReach::DynamicScalableGPUKReach(const GPUGraph &graph, distance_t k, uint32_t b1, uint32_t b2)
        : graph_(graph), k_(k), D1_(*this, graph, k, b1), D2_(*this, graph, k, b2, true),
          degree_(graph.num_vertices()), distance_(graph.num_vertices(), INF8), queue_(graph.num_vertices()),
          d_distance_(graph.num_vertices(), mgpu_context), d_visited_bits_(mgpu::div_up(graph.num_vertices(), 32), mgpu_context), 
          d_d1_bits_(mgpu::div_up(graph.num_vertices(), 32), mgpu_context), d_d1_(graph.num_vertices(), mgpu_context) {
    std::vector<std::pair<degree_t, vertex_t>> quedeg_temp_;
    quedeg_temp_.reserve(graph.num_vertices());
    DegreeQueue(less<pair<degree_t, vertex_t>>(), move(quedeg_temp_)).swap(quedeg_);
    updated_.reserve(graph.num_vertices());
    std::vector<std::pair<distance_t, vertex_t>> quedist_temp_;
    quedist_temp_.reserve(graph.num_vertices());
    DistanceQueue(greater<pair<distance_t, vertex_t>>(), move(quedist_temp_)).swap(quedist_);
}

void DynamicScalableGPUKReach::construct() {
    D1_.construct();
    D2_.construct();
}

bool DynamicScalableGPUKReach::query(vertex_t s, vertex_t t) const {
    if (s == t && s < graph_.num_vertices()) {
        return true;
    }
    if (D1_.indexed(s) && D1_.indexed(t)) {
        return D1_.distance(s, t) <= k_;
    }
    else if (D1_.indexed(s) || D1_.indexed(t)) {
        return D1_.distance(s, t) <= k_ || D1_.single_intermediate(s, t);
    }
    else if (D2_.indexed(s) && D2_.indexed(t)) {
        return D2_.distance(s, t) <= k_ || D1_.double_intermediate(s, t);
    }
    else if (D2_.indexed(s) || D2_.indexed(t)) {
        return D2_.distance(s, t) <= k_ || D2_.single_intermediate(s, t) || D1_.double_intermediate(s, t);
    }
    else {
        return D1_.double_intermediate(s, t) || D2_.double_intermediate(s, t) || bfs(s, t);
    }
}

bool DynamicScalableGPUKReach::bfs(vertex_t s, vertex_t t) const {
    back_ = 0;
    front_ = 0;
    distance_.at(s) = 0;
    queue_.at(back_++) = s;
    while (back_ != front_ && distance_.at(t) == INF8) {
        vertex_t cur = queue_.at(front_++);
        if (distance_.at(cur) >= k_) {
            continue;
        }
        for (const auto &nxt : graph_.successors(cur)) {
            if (distance_.at(nxt) == INF8 && !D1_.indexed(nxt) && !D2_.indexed(nxt)) {
                distance_.at(nxt) = distance_.at(cur) + 1;
                queue_.at(back_++) = nxt;
            }
        }
    }
    bool result = distance_.at(t) <= k_;
    for (size_t i = 0; i < back_; ++i) {
        distance_.at(queue_.at(i)) = INF8;
    }
    return result;
}

void
DynamicScalableGPUKReach::insert_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    D1_.insert_vertex(v, out, in);
    D2_.insert_vertex(v, out, in);
}
