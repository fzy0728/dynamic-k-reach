#include "hip/hip_runtime.h"
#include "ScalableGPUKReach.h"

using namespace std;

void gpu_bfs(const ScalableGPUKReach::PartialIndex &kReach, vertex_t s, const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
    const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
    distance_t* distance_data = kReach.d_distance_.data();
    int* visited_bits_data = kReach.d_visited_bits_.data();
    auto d1_bits_data = kReach.d_d1_bits_.data();
    auto isD2 = kReach.isD2_;

    hipMemset(distance_data, 0x3f, kReach.d_distance_.size() * sizeof(distance_t));
    hipMemset(visited_bits_data, 0, kReach.d_visited_bits_.size() * sizeof(int));

    mgpu::transform([=]MGPU_DEVICE(int index) {
    visited_bits_data[s / 32] = 1 << (31 & s);
    distance_data[s] = 0;
    }, 1, mgpu_context);

  workload_t wl;
  wl.count = h_vertices.at(s + 1) - h_vertices.at(s);
  wl.num_segments = 1;
  vector<int> edge_indices_host = { (int) h_vertices.at(s) };
  wl.segments = mgpu::fill<int>(0, 1, mgpu_context);
  wl.edge_indices = mgpu::to_mem(edge_indices_host, mgpu_context);
  for (int cur_level = 0; cur_level < kReach.k_ && wl.num_segments; ++cur_level) {
      // Create a dynamic work-creation engine.
    auto engine = mgpu::expt::lbs_workcreate(wl.count, wl.segments.data(), 
        wl.num_segments, mgpu_context);

    // The upsweep attempts atomicOr. If it succeeds, return the number of 
    // edges for that vertex.
    auto wl2_count = engine.upsweep(
        [=]MGPU_DEVICE(int index, int seg, int rank, mgpu::tuple<int> desc) {
        int count = 0;
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int mask = 1<< (31 & neighbor);
        if (0 == (mask & atomicOr(visited_bits_data + neighbor / 32, mask))) {
            if (!isD2 || !(mask & d1_bits_data[neighbor / 32])){
                count = vertices[neighbor + 1] - vertices[neighbor];
            }
            distance_data[neighbor] = cur_level + 1;
        }
        return count;
        }, mgpu::make_tuple(wl.edge_indices.data())
    );

    // The downsweep streams out the new edge pointers.
    mgpu::mem_t<int> edge_indices(wl2_count.num_segments, mgpu_context);
    mgpu::mem_t<int> segments = engine.downsweep(
        [=]MGPU_DEVICE(int dest_seg, int index, int seg, int rank, 
        mgpu::tuple<int> desc, int* out_edge_indices) {
        // Return the same count as before and store output segment-specific
        // data using dest_seg.
        int neighbor = edges[mgpu::get<0>(desc) + rank];
        int begin = vertices[neighbor];
        int end = vertices[neighbor + 1];

        // Store the pointer into the edges array for the new work segment.
        out_edge_indices[dest_seg] = begin;

        return end - begin;
        }, mgpu::make_tuple(wl.edge_indices.data()), edge_indices.data()
    );

    // Update the workload.
    wl.count = wl2_count.count;
    wl.num_segments = wl2_count.num_segments;
    wl.segments = std::move(segments);
    wl.edge_indices = std::move(edge_indices);
  }
  mgpu::dtoh(dist, distance_data, dist.size());
}

void gpu_set_d1(const ScalableGPUKReach::PartialIndex &kReach) {
    auto d1_bits_data = kReach.d_d1_bits_.data();
    auto d1_data = kReach.d_d1_.data();
    mgpu::htod(d1_data, kReach.queue_.data(), kReach.parent_.D1_.succ_.size());
    hipMemset(d1_bits_data, 0, kReach.d_d1_bits_.size() * sizeof(int));
    mgpu::transform([=]MGPU_DEVICE(int index) {
        vertex_t vertex = d1_data[index];
        atomicOr(d1_bits_data + vertex / 32, 1 << (31 & vertex));
    }, kReach.parent_.D1_.succ_.size(), mgpu_context);
}

ScalableGPUKReach::PartialIndex::PartialIndex(ScalableGPUKReach &parent,
                                           const GPUGraph &graph, distance_t k, uint32_t budget, bool isD2)
        : parent_(parent), graph_(graph), k_(k), budget_(budget), isD2_(isD2),
          quedeg_(parent.quedeg_), degree_(parent.degree_), queue_(parent.queue_), 
          d_distance_(parent.d_distance_), d_visited_bits_(parent.d_visited_bits_), d_d1_bits_(parent.d_d1_bits_), d_d1_(parent.d_d1_) {}

void ScalableGPUKReach::PartialIndex::construct() {
    succ_.clear();
    pred_.clear();
    set_degree();
    for (degree_t i = 0; i < graph_.num_vertices(); ++i) {
        quedeg_.emplace(degree_.at(i), i);
    }
    while (!quedeg_.empty() && succ_.size() != budget_) {
        auto deg = quedeg_.top().first;
        auto cur = quedeg_.top().second;
        quedeg_.pop();
        if (deg <= 0 || degree_.at(cur) <= 0) {
            continue;
        }
        if (deg != degree_.at(cur)) {
            quedeg_.emplace(degree_.at(cur), cur);
            continue;
        }
        succ_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
        construct_bfs(cur, graph_.vertices(), graph_.edges(), graph_.gpu_vertices(), graph_.gpu_edges(), succ_.at(cur));
        pred_[cur] = vector<distance_t>(graph_.num_vertices(), INF8);
        construct_bfs(cur, graph_.rvertices(), graph_.redges(), graph_.gpu_rvertices(), graph_.gpu_redges(), pred_.at(cur));
        update_cover(cur);
    }
    while (!quedeg_.empty()) {
        quedeg_.pop();
    }
}

void ScalableGPUKReach::PartialIndex::set_degree() {
    copy(graph_.degree().begin(), graph_.degree().end(), degree_.begin());
    if (isD2_) {
        size_t j = 0;
        for (const auto &i : parent_.D1_.succ_) {
            cover(i.first);
            queue_.at(j++) = i.first;
        }
        gpu_set_d1(*this);
    }
}

void ScalableGPUKReach::PartialIndex::cover(vertex_t v) {
    degree_.at(v) = 0;
    for (const auto &i : graph_.successors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
    for (const auto &i : graph_.predecessors(v)) {
        if (degree_.at(i) > 0) {
            --degree_.at(i);
        }
    }
}

void ScalableGPUKReach::PartialIndex::update_cover(vertex_t v) {
    for (vertex_t j = 0; j < graph_.num_vertices(); ++j) {
        if ((succ_.at(v).at(j) != INF8 || pred_.at(v).at(j) != INF8) &&
            degree_.at(j) > 0) {
            cover(j);
        }
    }
}

void ScalableGPUKReach::PartialIndex::construct_bfs(vertex_t s, const std::vector<degree_t> &h_vertices, const std::vector<vertex_t> &h_edges,
        const degree_t* vertices, const vertex_t* edges, std::vector<distance_t> &dist) {
   gpu_bfs(*this, s, h_vertices, h_edges, vertices, edges, dist);
}

bool ScalableGPUKReach::PartialIndex::indexed(vertex_t v) const {
    return succ_.find(v) != succ_.end();
}

distance_t ScalableGPUKReach::PartialIndex::distance(vertex_t s, vertex_t t) const {
    return indexed(s) ? succ_.at(s).at(t) : pred_.at(t).at(s);
}

bool ScalableGPUKReach::PartialIndex::single_intermediate(vertex_t s, vertex_t t) const {
    for (const auto &i : succ_) {
        if (pred_.at(i.first).at(s) + succ_.at(i.first).at(t) <= k_) {
            return true;
        }
    }
    return false;
}

bool ScalableGPUKReach::PartialIndex::double_intermediate(vertex_t s, vertex_t t) const {
    for (const auto &i : succ_) {
        for (const auto &j : succ_) {
            if (pred_.at(i.first).at(s) + succ_.at(i.first).at(j.first) + succ_.at(j.first).at(t) <= k_) {
                return true;
            }
        }
    }
    return false;
}

void ScalableGPUKReach::PartialIndex::insert_edge(vertex_t s, vertex_t t) {
    (void) s;
    (void) t;
    construct();
}

void ScalableGPUKReach::PartialIndex::remove_edge(vertex_t s, vertex_t t) {
    (void) s;
    (void) t;
    construct();
}

void ScalableGPUKReach::PartialIndex::remove_vertex(vertex_t v, const std::vector<vertex_t> &out,
                                                 const std::vector<vertex_t> &in) {
    (void) v;
    (void) out;
    (void) in;
    construct();
}

void ScalableGPUKReach::PartialIndex::insert_vertex(vertex_t v, const std::vector<vertex_t> &out,
                                                 const std::vector<vertex_t> &in) {
    (void) v;
    (void) out;
    (void) in;
    construct();
}


ScalableGPUKReach::ScalableGPUKReach(const GPUGraph &graph, distance_t k, uint32_t b1, uint32_t b2)
        : graph_(graph), k_(k), D1_(*this, graph, k, b1), D2_(*this, graph, k, b2, true),
          degree_(graph.num_vertices()), distance_(graph.num_vertices(), INF8), queue_(graph.num_vertices()), 
          d_distance_(graph.num_vertices(), mgpu_context), d_visited_bits_(mgpu::div_up(graph.num_vertices(), 32), mgpu_context), 
          d_d1_bits_(mgpu::div_up(graph.num_vertices(), 32), mgpu_context), d_d1_(graph.num_vertices(), mgpu_context) {
    vector<pair<degree_t, vertex_t>> quedeg_temp_;
    quedeg_temp_.reserve(graph.num_vertices());
    DegreeQueue(less<pair<degree_t, vertex_t>>(), move(quedeg_temp_)).swap(quedeg_);
}

void ScalableGPUKReach::construct() {
    D1_.construct();
    D2_.construct();
}

void ScalableGPUKReach::insert_edge(vertex_t s, vertex_t t) {
    (void) s;
    (void) t;
    construct();
}

void ScalableGPUKReach::remove_edge(vertex_t s, vertex_t t) {
    (void) s;
    (void) t;
    construct();
}

void ScalableGPUKReach::remove_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    (void) v;
    (void) out;
    (void) in;
    construct();
}

bool ScalableGPUKReach::query(vertex_t s, vertex_t t) const {
    if (s == t && s < graph_.num_vertices()) {
        return true;
    }
    if (D1_.indexed(s) && D1_.indexed(t)) {
        return D1_.distance(s, t) <= k_;
    }
    else if (D1_.indexed(s) || D1_.indexed(t)) {
        return D1_.distance(s, t) <= k_ || D1_.single_intermediate(s, t);
    }
    else if (D2_.indexed(s) && D2_.indexed(t)) {
        return D2_.distance(s, t) <= k_ || D1_.double_intermediate(s, t);
    }
    else if (D2_.indexed(s) || D2_.indexed(t)) {
        return D2_.distance(s, t) <= k_ || D2_.single_intermediate(s, t) || D1_.double_intermediate(s, t);
    }
    else {
        return D1_.double_intermediate(s, t) || D2_.double_intermediate(s, t) || bfs(s, t);
    }
}

bool ScalableGPUKReach::bfs(vertex_t s, vertex_t t) const {
    back_ = 0;
    front_ = 0;
    distance_.at(s) = 0;
    queue_.at(back_++) = s;
    while (back_ != front_ && distance_.at(t) == INF8) {
        vertex_t cur = queue_.at(front_++);
        if (distance_.at(cur) >= k_) {
            continue;
        }
        for (const auto &nxt : graph_.successors(cur)) {
            if (distance_.at(nxt) == INF8 && !D1_.indexed(nxt) && !D2_.indexed(nxt)) {
                distance_.at(nxt) = distance_.at(cur) + 1;
                queue_.at(back_++) = nxt;
            }
        }
    }
    bool result = distance_.at(t) <= k_;
    for (size_t i = 0; i < back_; ++i) {
        distance_.at(queue_.at(i)) = INF8;
    }
    return result;
}

void ScalableGPUKReach::insert_vertex(vertex_t v, const std::vector<vertex_t> &out, const std::vector<vertex_t> &in) {
    (void) v;
    (void) out;
    (void) in;
    construct();
}


